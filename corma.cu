#include "hip/hip_runtime.h"
#include <stdio.h>
#include <stdlib.h>
#include <math.h>
#include <string.h>

#define W 4000
#define H 20530

void fillrandom(float *matrix){
	int i,j;

	for(i = 0; i < H; i++){
		for(j = 0; j < W; j++){
			matrix[i * W + j] = drand48();
		}
	}
}

/**
 * Read tab delimited matrix file
 **/
int readmatrix(const char *filename, float *matrix){
	FILE *input;
	char *line = NULL;
	size_t size;
	char *token;
	int count = 0;

	input = fopen(filename, "r");
	if(input == NULL){
		return(-1);
	}

	while(getline(&line, &size, input) != -1){
		token = strtok(line, "\t");
		matrix[count] = atof(token);
		count++;
		while( (token = strtok(NULL, "\t")) != NULL){
			matrix[count] = atof(token);
			count++;
		}
	}

	fclose(input);
	free(line);
	return(0);
}

__global__ void calcmean(float *matrix, float *mean){

}

void calcmean(float *matrix, float *mean){
	int i,j;
	float sum;

	for(i = 0; i < H; i++){
		sum = 0.0;
		for(j = 0; j < W; j++){
			sum += matrix[i * W + j];
		}
		mean[i] = sum / (float)W;
	}
}

void calc_mm_std(float *matrix, float *mean, float *mm, float *std){
	int i,j;
	float sum, diff;

	for(i = 0; i < H; i++){
		sum = 0.0;
		for(j = 0; j < W; j++){
			diff = matrix[i * W + j] - mean[i];
			mm[i * W + j] = diff;
			sum += diff * diff;
		}
		std[i] = sqrtf(sum);
	}
}

__global__ void pearson(float *mm, float *std){
	int i, sample1, sample2;
	float sum,r;


	for(sample1 = 0; sample1 < H-1; sample1++){
		for(sample2 = sample1+1; sample2 < H; sample2++){
			sum = 0.0;
			for(i = 0; i < W; i++){
				sum += mm[sample1 * W + i] * mm[sample2 * W + i];
			}
			r = sum / (std[sample1] * std[sample2]);
		}
		printf("%d\n", sample1);
	}
}

int main(int argc, char **argv){
	float *matrix, *minusmean, *mean, *std;

	hipMallocManaged(&matrix, sizeof(float) * W * H);

	if(matrix == NULL){
		return(1);
	}

	hipMallocManaged(&minusmean, sizeof(float) * W * H);
	if(minusmean == NULL){
		return(1);
	}

	hipMallocManaged(&mean, sizeof(float) * H);
	hipMallocManaged(&std, sizeof(float) * H);
	if(mean == NULL || std == NULL){
		return(1);
	}

	if(argc != 2){
		printf("Missing matrix file\n");
	}
	else{
		readmatrix(argv[1], matrix);
		calcmean(matrix, mean);
		//calc_mm_std(matrix, mean, minusmean, std);
		//pearson(minusmean, std);
	}

	hipFree(mean);
	hipFree(std);
	hipFree(matrix);
	hipFree(minusmean);
	return(EXIT_SUCCESS);
}
